#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "debug.h"
#include "timer.h"
#include "utils_cuda.h"
#include "utils_file.h"

#include "params.h"


#define WARP 32


int device=0;

__device__ __inline__ float2 Get_W_value(int N, int m){
	float2 ctemp;
	ctemp.x=-cosf( 6.283185f*fdividef( (float) m, (float) N ) - 3.141592654f );
	ctemp.y=sinf( 6.283185f*fdividef( (float) m, (float) N ) - 3.141592654f );
	return(ctemp);
}

__device__ __inline__ float2 Get_W_value_float(float N, float m){
	float2 ctemp;
	ctemp.x=-cosf( 6.283185f*fdividef( m, N) - 3.141592654f );
	ctemp.y=sinf( 6.283185f*fdividef( m, N) - 3.141592654f );
	return(ctemp);
}


__device__ void do_FFT(float2 *s_input, int N, int bits){ // in-place
	float2 A_DFT_value, B_DFT_value, ftemp2, ftemp;
	float2 WB;
	
	int r, j, k, PoTm1, A_read_index, B_read_index, A_write_index, B_write_index, Nhalf;
	int An, A_load_id;

	Nhalf=N>>1;
	
	//-----------------------------------------------
	//----- First N-1 iteration
	PoTm1=1;
	
	A_read_index=threadIdx.x;
	B_read_index=threadIdx.x + Nhalf;
		
	A_write_index=2*threadIdx.x;
	B_write_index=2*threadIdx.x+1;

	A_load_id = 2*threadIdx.x;
	An=2*threadIdx.x;
	
	for(r=1;r<bits;r++){
		An >>= 1;
		A_load_id <<= 1;
		A_load_id |= An & 1;
		
		j=(threadIdx.x)>>(r-1);

		k=PoTm1*j;
		
		ftemp  = s_input[A_read_index];
		ftemp2 = s_input[B_read_index];
		
		A_DFT_value.x=ftemp.x + ftemp2.x;
		A_DFT_value.y=ftemp.y + ftemp2.y;
		
		WB = Get_W_value(N,k);
		
		B_DFT_value.x=WB.x*(ftemp.x - ftemp2.x) - WB.y*(ftemp.y - ftemp2.y);
		B_DFT_value.y=WB.x*(ftemp.y - ftemp2.y) + WB.y*(ftemp.x - ftemp2.x);
		
		PoTm1=PoTm1<<1;
		
		__syncthreads();
		s_input[A_write_index]=A_DFT_value;
		s_input[B_write_index]=B_DFT_value;
		__syncthreads();
	}
	A_load_id &= N-1;
	//----- First N-1 iteration
	//-----------------------------------------------
	
	
	//-----------------------------------------------
	//----- Last exchange
	ftemp  = s_input[A_read_index];
	ftemp2 = s_input[B_read_index];
	
	A_DFT_value.x = ftemp.x + ftemp2.x;
	A_DFT_value.y = ftemp.y + ftemp2.y;
	B_DFT_value.x = ftemp.x - ftemp2.x;
	B_DFT_value.y = ftemp.y - ftemp2.y;
	
	__syncthreads();
	s_input[A_write_index]=A_DFT_value;
	s_input[B_write_index]=B_DFT_value;
	__syncthreads();
	//----- Last exchange
	//-----------------------------------------------
	
	//-----------------------------------------------
	//----- De-shuffle
	ftemp=s_input[A_load_id];
	ftemp2=s_input[A_load_id+Nhalf];
	__syncthreads();
	s_input[2*threadIdx.x]=ftemp;
	s_input[2*threadIdx.x+1]=ftemp2;
	//----- De-shuffle
	//-----------------------------------------------
	
	
	/*
	//-----------------------------------------------
	//----- Last exchange + De-shuffle
		ftemp  = s_input[A_load_id];
		ftemp2 = s_input[A_load_id+Nhalf];
		
		A_DFT_value.x = ftemp.x + ftemp2.x;
		A_DFT_value.y = ftemp.y + ftemp2.y;
		B_DFT_value.x = ftemp.x - ftemp2.x;
		B_DFT_value.y = ftemp.y - ftemp2.y;
		
		__syncthreads();
		s_input[threadIdx.x]=A_DFT_value;
		s_input[threadIdx.x+Nhalf]=B_DFT_value;
		__syncthreads();
	//----- Last exchange
	//-----------------------------------------------	
	*/
		
	//-------> END
}

__global__ void FFT_GPU_external(float2 *d_input, float2* d_output, int N, int bits) {
	extern __shared__ float2 s_input[];
	s_input[threadIdx.x]=d_input[threadIdx.x + blockIdx.x*N];
	s_input[threadIdx.x + N/2]=d_input[threadIdx.x + N/2 + blockIdx.x*N];
	
	__syncthreads();
	do_FFT(s_input,N,bits);
	
	__syncthreads();
	d_output[threadIdx.x + blockIdx.x*N]=s_input[threadIdx.x];
	d_output[threadIdx.x + N/2 + blockIdx.x*N]=s_input[threadIdx.x + N/2];
}

__global__ void FFT_GPU_multiple(float2 *d_input, float2* d_output, int N, int bits) {
	extern __shared__ float2 s_input[];
	s_input[threadIdx.x]=d_input[threadIdx.x + blockIdx.x*N];
	s_input[threadIdx.x + N/2]=d_input[threadIdx.x + N/2 + blockIdx.x*N];
	
	__syncthreads();
	for(int f=0;f<100;f++){
		do_FFT(s_input,N,bits);
	}
	
	__syncthreads();
	d_output[threadIdx.x + blockIdx.x*N]=s_input[threadIdx.x];
	d_output[threadIdx.x + N/2 + blockIdx.x*N]=s_input[threadIdx.x + N/2];
}

//*****************************************************************************
//*****************************************************************************
//*****************************************************************************

int Max_columns_in_memory_shared(int nSamples, int nSpectra) {
	long int nColumns,maxgrid_x;

	size_t free_mem,total_mem;
	hipDeviceProp_t devProp;
	
	checkCudaErrors(hipSetDevice(device));
	checkCudaErrors(hipGetDeviceProperties(&devProp,device));
	maxgrid_x = devProp.maxGridSize[0];
	hipMemGetInfo(&free_mem,&total_mem);
	
	nColumns=((long int) free_mem)/(2.0*sizeof(float2)*nSamples);
	if(nColumns>maxgrid_x) nColumns=maxgrid_x;
	nColumns=(int) nColumns*0.9;
	return(nColumns);
}


void FFT_init(){
	//---------> Specific nVidia stuff
	hipDeviceSetCacheConfig(hipFuncCachePreferEqual);
	hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
}

void FFT_external_benchmark(float2 *d_input, float2 *d_output, int nSamples, int nSpectra, double *FFT_time){
	GpuTimer timer;
	//---------> CUDA block and CUDA grid parameters
	int nCUDAblocks_x=nSpectra;
	int nCUDAblocks_y=1; //Head size
	
	dim3 gridSize(nCUDAblocks_x, nCUDAblocks_y, 1);	//nCUDAblocks_y goes through spectra
	dim3 blockSize(nSamples/2, 1, 1); 				//nCUDAblocks_x goes through channels
	
	//---------> FIR filter part
	timer.Start();
	FFT_GPU_external<<<gridSize, blockSize,nSamples*8>>>( d_input, d_output, nSamples,round(log(nSamples)/log(2)));
	timer.Stop();
	*FFT_time += timer.Elapsed();
}

void FFT_multiple_benchmark(float2 *d_input, float2 *d_output, int nSamples, int nSpectra, double *FFT_time){
	GpuTimer timer;
	//---------> CUDA block and CUDA grid parameters
	dim3 gridSize_multiple(1000, 1, 1);	//nCUDAblocks_y goes through spectra
	dim3 blockSize(nSamples/2, 1, 1); 				//nCUDAblocks_x goes through channels
	
	//---------> FIR filter part
	timer.Start();
	FFT_GPU_multiple<<<gridSize_multiple, blockSize,nSamples*8>>>( d_input, d_output, nSamples,round(log(nSamples)/log(2)));
	timer.Stop();
	*FFT_time += timer.Elapsed();
}


//*****************************************************************************
//*****************************************************************************
//*****************************************************************************


int GPU_FFT(float2 *h_input, float2 *h_output, int nSamples, int nSpectra, int inverse){
	//---------> Initial nVidia stuff
	int devCount;
	size_t free_mem,total_mem;
	checkCudaErrors(hipGetDeviceCount(&devCount));
	checkCudaErrors(hipSetDevice(device));
	
	hipMemGetInfo(&free_mem,&total_mem);
	if(DEBUG) printf("\nDevice has %ld MB of total memory, which %ld MB is available.\n", (long int) total_mem/(1000*1000), (long int) free_mem/(1000*1000));
	
	//---------> Checking memory
	int nElements=nSamples*nSpectra;
	int input_size=nElements;
	int output_size=nElements;
	
	float free_memory = (float) free_mem/(1024.0*1024.0);
	float memory_required=((2*input_size + 2*output_size)*sizeof(float))/(1024.0*1024.0);
	if(DEBUG) printf("DEBUG: Device has %0.3f MB of total memory, which %0.3f MB is available. Memory required %0.3f MB\n", (float) total_mem/(1024.0*1024.0), free_memory ,memory_required);
	if(memory_required>free_memory) {printf("\n \n Array is too big for the device! \n \n"); return(-3);}
		
	//---------> Measurements
	double transfer_in, transfer_out, FFT_time, FFT_external_time, FFT_multiple_time, FFT_multiple_reuse_time,cuFFT_time,FFT_multiple_reuse_registers_time;
	GpuTimer timer; // if set before set device getting errors - invalid handle  
	
	
	//------------------------------------------------------------------------------
	//---------> Shared memory kernel
	transfer_in=0.0; transfer_out=0.0; FFT_time=0.0; FFT_external_time=0.0; FFT_multiple_time=0.0; FFT_multiple_reuse_time=0.0; cuFFT_time=0.0; FFT_multiple_reuse_registers_time=0.0;

	//---------> Memory allocation
	if (DEBUG) printf("Device memory allocation...: \t\t");
	float2 *d_output;
	float2 *d_input;
	timer.Start();
	checkCudaErrors(hipMalloc((void **) &d_input,  sizeof(float2)*input_size));
	checkCudaErrors(hipMalloc((void **) &d_output, sizeof(float2)*output_size));
	timer.Stop();
	if (DEBUG) printf("done in %g ms.\n", timer.Elapsed());

	//---------> FFT calculation
	if (DEBUG) printf("Transferring data to device...: \t");
	timer.Start();
	checkCudaErrors(hipMemcpy(d_input, h_input, input_size*sizeof(float2), hipMemcpyHostToDevice));
	timer.Stop();
	transfer_in+=timer.Elapsed();
	if (DEBUG) printf("done in %g ms.\n", timer.Elapsed());

	//-----> Compute FFT on the chunk
	if(CUFFT){
		//---------> FFT
		hipfftHandle plan;
		hipfftResult error;
		error = hipfftPlan1d(&plan, nSamples, HIPFFT_C2C, nSpectra);
		if (HIPFFT_SUCCESS != error){
			printf("CUFFT error: %d", error);
		}
		
		timer.Start();
		hipfftExecC2C(plan, (hipfftComplex *)d_input, (hipfftComplex *)d_output, HIPFFT_FORWARD);
		timer.Stop();
		cuFFT_time += timer.Elapsed();
		
		hipfftDestroy(plan);
	}
	
	if(MULTIPLE){
		if (DEBUG) printf("Multiple FFT...: \t\t\t");
		FFT_init();
		FFT_multiple_benchmark(d_input, d_output, nSamples, nSpectra, &FFT_multiple_time);
		if (DEBUG) printf("done in %g ms.\n", FFT_multiple_time);
	}

	if(EXTERNAL){
		if (DEBUG) printf("FFT...: \t\t\t\t");
		FFT_init();
		FFT_external_benchmark(d_input, d_output, nSamples, nSpectra, &FFT_external_time);
		if (DEBUG) printf("done in %g ms.\n", FFT_external_time);
	}
	
	//-----> Copy chunk of output data to host
	if (DEBUG) printf("Transferring data to host...: \t\t");
	timer.Start();
	checkCudaErrors(hipMemcpy( h_output, d_output, output_size*sizeof(float2), hipMemcpyDeviceToHost));
	timer.Stop();
	transfer_out+=timer.Elapsed();
	if (DEBUG) printf("done in %g ms.\n", timer.Elapsed());

	

	//---------> error check -----
	checkCudaErrors(hipGetLastError());
	
	//---------> Feeing allocated resources
	checkCudaErrors(hipFree(d_input));
	checkCudaErrors(hipFree(d_output));
	
	if (DEBUG || WRITE) printf("nSpectra:%d; nSamples:%d cuFFT:%0.3f ms; FFT:%0.3f ms; FFT external:%0.3f ms; FFT multiple:%0.3f ms; FFT multiple reuse:%0.3f ms; FFT_multiple_reuse_registers_time:%0.3fms; HostToDevice:%0.3f ms; DeviceToHost:%0.3f ms\n",nSpectra,nSamples,cuFFT_time, FFT_time, FFT_external_time, FFT_multiple_time, FFT_multiple_reuse_time, FFT_multiple_reuse_registers_time, transfer_in, transfer_out);	
	
	if (WRITE){ 
		char str[200];
		sprintf(str,"GPU-FFT-Pease.dat");
		if (DEBUG) printf("\n Write results into file...\t");
		save_time(str, nSpectra,nSamples, cuFFT_time, FFT_time, FFT_external_time, FFT_multiple_time, FFT_multiple_reuse_time, FFT_multiple_reuse_registers_time, transfer_in, transfer_out);
		if (DEBUG) printf("\t done.\n-------------------------------------\n");
	}
	
	return(1);
}