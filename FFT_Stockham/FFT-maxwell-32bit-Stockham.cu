#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "debug.h"
#include "timer.h"
#include "utils_cuda.h"
#include "utils_file.h"

#include "params.h"


#define WARP 32


int device=0;

__device__ __inline__ float2 Get_W_value(int N, int m){
	float2 ctemp;
	ctemp.x=cosf( -2.0f*3.141592654f*fdividef( (float) m, (float) N) );
	ctemp.y=sinf( -2.0f*3.141592654f*fdividef( (float) m, (float) N) );
	return(ctemp);
}


__device__ void do_FFT(float2 *s_input, int N, int bits){
	float2 DFT_value_even, DFT_value_odd, ftemp2, ftemp;
	float2 W;
	
	int r, j, k, PoT, PoTm1, A_index, B_index, Nhalf;

	Nhalf=N>>1;
	
	//-----> FFT
	//--> 
	
	PoT=1;
	PoTm1=0;
	//------------------------------------------------------------
	// First iteration
		PoTm1=PoT;
		PoT=PoT<<1;
		
		j=threadIdx.x;
		k=0;
		
		W.x=1;
		W.y=0;

		A_index=j*PoTm1;
		B_index=j*PoTm1 + Nhalf;
		
		ftemp2=s_input[B_index];
		ftemp=s_input[A_index];
		
		DFT_value_even.x=ftemp.x + W.x*ftemp2.x - W.y*ftemp2.y;
		DFT_value_even.y=ftemp.y + W.x*ftemp2.y + W.y*ftemp2.x;
		
		DFT_value_odd.x=ftemp.x - W.x*ftemp2.x + W.y*ftemp2.y;
		DFT_value_odd.y=ftemp.y - W.x*ftemp2.y - W.y*ftemp2.x;
		
		__syncthreads();
		s_input[j*PoT]=DFT_value_even;
		s_input[j*PoT + PoTm1]=DFT_value_odd;
		__syncthreads();
	// First iteration
	//------------------------------------------------------------
	
	for(r=2;r<=bits;r++){
		PoTm1=PoT;
		PoT=PoT<<1;
		
		j=threadIdx.x>>(r-1);
		k=threadIdx.x & (PoTm1-1);
		
		W=Get_W_value(PoT,k);

		A_index=j*PoTm1 + k;
		B_index=j*PoTm1 + k + Nhalf;
		
		ftemp2=s_input[B_index];
		ftemp=s_input[A_index];
		
		DFT_value_even.x=ftemp.x + W.x*ftemp2.x - W.y*ftemp2.y;
		DFT_value_even.y=ftemp.y + W.x*ftemp2.y + W.y*ftemp2.x;
		
		DFT_value_odd.x=ftemp.x - W.x*ftemp2.x + W.y*ftemp2.y;
		DFT_value_odd.y=ftemp.y - W.x*ftemp2.y - W.y*ftemp2.x;
		
		__syncthreads();
		s_input[j*PoT + k]=DFT_value_even;
		s_input[j*PoT + k + PoTm1]=DFT_value_odd;
		__syncthreads();
	}
	//-------> END
}


__device__ void do_FFT_outofplace(float2 *s_input, float2 *s_output, int N, int bits){
	float2 DFT_value_even, DFT_value_odd, ftemp2, ftemp;
	float2 W;
	
	int r, j, k, PoT, PoTm1, A_index, B_index, Nhalf;

	Nhalf=N>>1;
	
	//-----> FFT
	//--> 
	
	PoT=1;
	PoTm1=0;
	for(r=1;r<=bits;r++){
		PoTm1=PoT;
		PoT=PoT<<1;
		
		j=threadIdx.x>>(r-1);
		k=threadIdx.x & (PoTm1-1);
		
		W=Get_W_value(PoT,k);

		A_index=j*PoTm1+k;
		B_index=j*PoTm1+k+Nhalf;
		
		ftemp2=s_input[B_index];
		ftemp=s_input[A_index];
		
		DFT_value_even.x=ftemp.x + W.x*ftemp2.x - W.y*ftemp2.y;
		DFT_value_even.y=ftemp.y + W.x*ftemp2.y + W.y*ftemp2.x;
		
		DFT_value_odd.x=ftemp.x - W.x*ftemp2.x + W.y*ftemp2.y;
		DFT_value_odd.y=ftemp.y - W.x*ftemp2.y - W.y*ftemp2.x;
		
		
		s_output[j*PoT + k]=DFT_value_even;
		s_output[j*PoT + k + PoTm1]=DFT_value_odd;
		
		__syncthreads();
		

		if((r+1)<=bits){
			r++;
			
			PoTm1=PoT;
			PoT=PoT<<1;
			
			j=threadIdx.x>>(r-1);
			k=threadIdx.x & (PoTm1-1);
			
			W=Get_W_value(PoT,k);

			A_index=j*PoTm1+k;
			B_index=j*PoTm1+k+Nhalf;
			
			ftemp2=s_output[B_index];
			ftemp=s_output[A_index];
			
			DFT_value_even.x=ftemp.x + W.x*ftemp2.x - W.y*ftemp2.y;
			DFT_value_even.y=ftemp.y + W.x*ftemp2.y + W.y*ftemp2.x;
			
			DFT_value_odd.x=ftemp.x - W.x*ftemp2.x + W.y*ftemp2.y;
			DFT_value_odd.y=ftemp.y - W.x*ftemp2.y - W.y*ftemp2.x;
			
			
			s_input[j*PoT + k]=DFT_value_even;
			s_input[j*PoT + k + PoTm1]=DFT_value_odd;
			__syncthreads();
		}	
	}
	
	if(r%2) {
		s_output[threadIdx.x]=s_input[threadIdx.x];
		s_output[threadIdx.x+Nhalf]=s_input[threadIdx.x+Nhalf];
	}
	//-------> END
}


__device__ void do_FFT_reuse(float2 *s_input, float2 *s_twidle, int N, int bits){
	float2 DFT_value_even, DFT_value_odd, ftemp2, ftemp;
	float2 W;
	
	int r, j, k, PoT, PoTm1, A_index, B_index, Nhalf;

	Nhalf=N>>1;
	
	//-----> FFT
	//--> 
	
	PoT=1;
	PoTm1=0;
	for(r=1;r<=bits;r++){
		PoTm1=PoT;
		PoT=PoT<<1;
		
		j=threadIdx.x>>(r-1);
		k=threadIdx.x & (PoTm1-1);
		
		W=s_twidle[(k*PoT)%N];

		A_index=j*PoTm1+k;
		B_index=j*PoTm1+k+Nhalf;
		
		ftemp2=s_input[B_index];
		ftemp=s_input[A_index];
		
		DFT_value_even.x=ftemp.x + W.x*ftemp2.x - W.y*ftemp2.y;
		DFT_value_even.y=ftemp.y + W.x*ftemp2.y + W.y*ftemp2.x;
		
		DFT_value_odd.x=ftemp.x - W.x*ftemp2.x + W.y*ftemp2.y;
		DFT_value_odd.y=ftemp.y - W.x*ftemp2.y - W.y*ftemp2.x;
		
		__syncthreads();
		s_input[j*PoT + k]=DFT_value_even;
		s_input[j*PoT + k + PoTm1]=DFT_value_odd;
		__syncthreads();
	}
	//-------> END
}


__device__ void do_FFT_reuse_registers(float2 *s_input, float2 *r_twiddle, int N){
	float2 DFT_value_even, DFT_value_odd, ftemp2, ftemp;
	float2 W;
	
	int r, j, k, PoT, PoTm1, A_index, B_index, Nhalf;

	Nhalf=N>>1;
	
	//-----> FFT
	//--> 
	
	PoT=1;
	PoTm1=0;
	for(r=1;r<=NBITS;r++){
		PoTm1=PoT;
		PoT=PoT<<1;
		
		j=threadIdx.x>>(r-1);
		k=threadIdx.x & (PoTm1-1);
		
		W=r_twiddle[r];

		A_index=j*PoTm1+k;
		B_index=j*PoTm1+k+Nhalf;
		
		ftemp2=s_input[B_index];
		ftemp=s_input[A_index];
		
		DFT_value_even.x=ftemp.x + W.x*ftemp2.x - W.y*ftemp2.y;
		DFT_value_even.y=ftemp.y + W.x*ftemp2.y + W.y*ftemp2.x;
		
		DFT_value_odd.x=ftemp.x - W.x*ftemp2.x + W.y*ftemp2.y;
		DFT_value_odd.y=ftemp.y - W.x*ftemp2.y - W.y*ftemp2.x;
		
		__syncthreads();
		s_input[j*PoT + k]=DFT_value_even;
		s_input[j*PoT + k + PoTm1]=DFT_value_odd;
		__syncthreads();
	}
	//-------> END
}



__global__ void FFT_GPU_external(float2 *d_input, float2* d_output, int N, int bits) {
	extern __shared__ float2 s_input[];
	s_input[threadIdx.x]=d_input[threadIdx.x + blockIdx.x*N];
	s_input[threadIdx.x + N/2]=d_input[threadIdx.x + N/2 + blockIdx.x*N];
	
	__syncthreads();
	do_FFT(s_input,N,bits);
	
	__syncthreads();
	d_output[threadIdx.x + blockIdx.x*N]=s_input[threadIdx.x];
	d_output[threadIdx.x + N/2 + blockIdx.x*N]=s_input[threadIdx.x + N/2];
}

__global__ void FFT_GPU_external_outofplace(float2 *d_input, float2* d_output, int N, int bits) {
	extern __shared__ float2 s_input_output[];
	s_input_output[threadIdx.x]=d_input[threadIdx.x + blockIdx.x*N];
	s_input_output[threadIdx.x + N/2]=d_input[threadIdx.x + N/2 + blockIdx.x*N];
	
	__syncthreads();
	do_FFT_outofplace(s_input_output,&s_input_output[N],N,bits);
	
	__syncthreads();
	d_output[threadIdx.x + blockIdx.x*N]=s_input_output[threadIdx.x + N];
	d_output[threadIdx.x + N/2 + blockIdx.x*N]=s_input_output[threadIdx.x + N/2 + N];
}


__global__ void FFT_GPU_multiple(float2 *d_input, float2* d_output, int N, int bits) {
	extern __shared__ float2 s_input[];
	s_input[threadIdx.x]=d_input[threadIdx.x + blockIdx.x*N];
	s_input[threadIdx.x + N/2]=d_input[threadIdx.x + N/2 + blockIdx.x*N];
	
	__syncthreads();
	for(int f=0;f<100;f++){
		do_FFT(s_input,N,bits);
	}
	
	__syncthreads();
	d_output[threadIdx.x + blockIdx.x*N]=s_input[threadIdx.x];
	d_output[threadIdx.x + N/2 + blockIdx.x*N]=s_input[threadIdx.x + N/2];
}


__global__ void FFT_GPU_multiple_reuse(float2 *d_input, float2* d_output, int N, int bits) {
	extern __shared__ float2 s_input_and_twidle[];
	s_input_and_twidle[threadIdx.x]=d_input[threadIdx.x + blockIdx.x*N];
	s_input_and_twidle[threadIdx.x + N/2]=d_input[threadIdx.x + N/2 + blockIdx.x*N];
	
	s_input_and_twidle[threadIdx.x + N]=Get_W_value(N,threadIdx.x);
	s_input_and_twidle[threadIdx.x + N + N/2]=Get_W_value(N,threadIdx.x + N/2);
	
	__syncthreads();
	for(int f=0;f<100;f++){
		do_FFT_reuse(s_input_and_twidle,&s_input_and_twidle[N],N,bits);
	}
	
	__syncthreads();
	d_output[threadIdx.x + blockIdx.x*N]=s_input_and_twidle[threadIdx.x];
	d_output[threadIdx.x + N/2 + blockIdx.x*N]=s_input_and_twidle[threadIdx.x + N/2];
}

__global__ void FFT_GPU_multiple_reuse_register(float2 *d_input, float2* d_output, int N, int bits) {
	extern __shared__ float2 s_input[];
	float2 r_twiddle[NBITS];
	int r, PoT, PoTm1, k;
	
	//---> Loading input data
	s_input[threadIdx.x]=d_input[threadIdx.x + blockIdx.x*N];
	s_input[threadIdx.x + N/2]=d_input[threadIdx.x + N/2 + blockIdx.x*N];
	
	//---> Calculating twiddle factors
	PoT=1;
	PoTm1=0;
	for(r=1;r<=NBITS;r++){
		PoTm1=PoT;
		PoT=PoT<<1;
		k=threadIdx.x & (PoTm1-1);
		
		r_twiddle[r]=Get_W_value(PoT,k);
	}
	
	__syncthreads();
	for(int f=0;f<100;f++){
		do_FFT_reuse_registers(s_input,r_twiddle,N);
	}
	
	__syncthreads();
	d_output[threadIdx.x + blockIdx.x*N]=s_input[threadIdx.x];
	d_output[threadIdx.x + N/2 + blockIdx.x*N]=s_input[threadIdx.x + N/2];
}

__global__ void FFT_GPU(float2 *d_input, float2* d_output, int N, int bits) {
	extern __shared__ float2 s_input[];
	// ----------> Load phase
	s_input[threadIdx.x]=d_input[threadIdx.x + blockIdx.x*N];
	s_input[threadIdx.x + N/2]=d_input[threadIdx.x + N/2 + blockIdx.x*N];
	
	__syncthreads();
	// ----------> FFT
	float2 DFT_value_even, DFT_value_odd, ftemp2, ftemp;
	float2 W;
	
	int r, j, k, PoT, PoTm1, A_index, B_index, Nhalf;

	Nhalf=N>>1;
	
	//-----> FFT
	//--> 
	
	A_index=threadIdx.x;
	B_index=threadIdx.x + Nhalf;
	
	PoT=1;
	PoTm1=0;
	for(r=1;r<=bits;r++){
		PoTm1=PoT;
		PoT=PoT<<1;
		
		j=threadIdx.x>>(r-1);
		k=threadIdx.x & (PoTm1-1);
		
		W=Get_W_value(PoT,k);

		//A_index=j*PoTm1+k;
		//B_index=j*PoTm1+k+Nhalf;
		
		ftemp2=s_input[B_index];
		ftemp=s_input[A_index];
		
		//printf("thread:%d; j:%d; k:%d; Writes:%d and %d; Reads:%d and %d\n", threadIdx.x, j, k, (j*PoT + k), (j*PoT + k + PoTm1), A_index, B_index);
		
		DFT_value_even.x=ftemp.x + W.x*ftemp2.x - W.y*ftemp2.y;
		DFT_value_even.y=ftemp.y + W.x*ftemp2.y + W.y*ftemp2.x;
		
		DFT_value_odd.x=ftemp.x - W.x*ftemp2.x + W.y*ftemp2.y;
		DFT_value_odd.y=ftemp.y - W.x*ftemp2.y - W.y*ftemp2.x;
		
		__syncthreads();
		s_input[j*PoT + k]=DFT_value_even;
		s_input[j*PoT + k + PoTm1]=DFT_value_odd;
		__syncthreads();
	}
	
	// ----------> Save phase
	d_output[threadIdx.x + blockIdx.x*N]=s_input[threadIdx.x];
	d_output[threadIdx.x + N/2 + blockIdx.x*N]=s_input[threadIdx.x + N/2];
}


int Max_columns_in_memory_shared(int nSamples, int nSpectra) {
	long int nColumns,maxgrid_x;

	size_t free_mem,total_mem;
	hipDeviceProp_t devProp;
	
	checkCudaErrors(hipSetDevice(device));
	checkCudaErrors(hipGetDeviceProperties(&devProp,device));
	maxgrid_x = devProp.maxGridSize[0];
	hipMemGetInfo(&free_mem,&total_mem);
	
	nColumns=((long int) free_mem)/(2.0*sizeof(float2)*nSamples);
	if(nColumns>maxgrid_x) nColumns=maxgrid_x;
	nColumns=(int) nColumns*0.9;
	return(nColumns);
}


void FFT_init(){
	//---------> Specific nVidia stuff
	hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
}

void FFT_benchmark(float2 *d_input, float2 *d_output, int nSamples, int nSpectra, double *FFT_time){
	const int multiple=1;
	GpuTimer timer;
	//---------> CUDA block and CUDA grid parameters
	int nCUDAblocks_x=nSpectra/multiple;
	int nCUDAblocks_y=1;
	
	dim3 gridSize(nCUDAblocks_x, nCUDAblocks_y, 1);
	dim3 blockSize(nSamples/2, 1, 1);
	
	//---------> FIR filter part
	timer.Start();
	FFT_GPU<<<gridSize, blockSize, nSamples*8>>>( d_input, d_output, nSamples,round(log(nSamples)/log(2))); //8->4
	timer.Stop();
	*FFT_time += timer.Elapsed();
}

void FFT_external_benchmark(float2 *d_input, float2 *d_output, int nSamples, int nSpectra, double *FFT_time){
	GpuTimer timer;
	//---------> CUDA block and CUDA grid parameters
	int nCUDAblocks_x=nSpectra;
	int nCUDAblocks_y=1;
	
	dim3 gridSize(nCUDAblocks_x, nCUDAblocks_y, 1);
	dim3 blockSize(nSamples/2, 1, 1);
	
	//---------> FIR filter part
	timer.Start();
	FFT_GPU_external<<<gridSize, blockSize,nSamples*8>>>( d_input, d_output, nSamples,round(log(nSamples)/log(2)));
	//FFT_GPU_external_outofplace<<<gridSize, blockSize,nSamples*8*2>>>( d_input, d_output, nSamples,round(log(nSamples)/log(2)));
	timer.Stop();
	*FFT_time += timer.Elapsed();
}

void FFT_multiple_benchmark(float2 *d_input, float2 *d_output, int nSamples, int nSpectra, double *FFT_time){
	GpuTimer timer;
	//---------> CUDA block and CUDA grid parameters
	dim3 gridSize_multiple(1000, 1, 1);
	dim3 blockSize(nSamples/2, 1, 1);
	
	//---------> FIR filter part
	timer.Start();
	FFT_GPU_multiple<<<gridSize_multiple, blockSize,nSamples*8>>>( d_input, d_output, nSamples,round(log(nSamples)/log(2)));
	timer.Stop();
	*FFT_time += timer.Elapsed();
}

void FFT_multiple_reuse_benchmark(float2 *d_input, float2 *d_output, int nSamples, int nSpectra, double *FFT_time){
	GpuTimer timer;
	//---------> CUDA block and CUDA grid parameters
	dim3 gridSize_multiple(1000, 1, 1);
	dim3 blockSize(nSamples/2, 1, 1);
	
	//---------> FIR filter part
	timer.Start();
	FFT_GPU_multiple_reuse<<<gridSize_multiple, blockSize,nSamples*8*2>>>( d_input, d_output, nSamples,round(log(nSamples)/log(2)));
	timer.Stop();
	*FFT_time += timer.Elapsed();
}

void FFT_multiple_reuse_registers_benchmark(float2 *d_input, float2 *d_output, int nSamples, int nSpectra, double *FFT_time){
	GpuTimer timer;
	//---------> CUDA block and CUDA grid parameters
	dim3 gridSize_multiple(1000, 1, 1);
	dim3 blockSize(nSamples, 1, 1);
	
	//---------> FIR filter part
	timer.Start();
	FFT_GPU_multiple_reuse_register<<<gridSize_multiple, blockSize,nSamples*8*2>>>( d_input, d_output, nSamples,round(log(nSamples)/log(2)));
	timer.Stop();
	*FFT_time += timer.Elapsed();
}

int GPU_FFT(float2 *h_input, float2 *h_output, int nSamples, int nSpectra, int inverse){
	//---------> Initial nVidia stuff
	int devCount;
	size_t free_mem,total_mem;
	checkCudaErrors(hipGetDeviceCount(&devCount));
	checkCudaErrors(hipSetDevice(device));
	
	hipMemGetInfo(&free_mem,&total_mem);
	if(DEBUG) printf("\nDevice has %ld MB of total memory, which %ld MB is available.\n", (long int) total_mem/(1000*1000), (long int) free_mem/(1000*1000));
	
	//---------> Checking memory
	int nElements=nSamples*nSpectra;
	int input_size=nElements;
	int output_size=nElements;
	
	float free_memory = (float) free_mem/(1024.0*1024.0);
	float memory_required=((2*input_size + 2*output_size)*sizeof(float))/(1024.0*1024.0);
	if(DEBUG) printf("DEBUG: Device has %0.3f MB of total memory, which %0.3f MB is available. Memory required %0.3f MB\n", (float) total_mem/(1024.0*1024.0), free_memory ,memory_required);
	if(memory_required>free_memory) {printf("\n \n Array is too big for the device! \n \n"); return(-3);}
		
	//---------> Measurements
	double transfer_in, transfer_out, FFT_time, FFT_external_time, FFT_multiple_time, FFT_multiple_reuse_time,cuFFT_time,FFT_multiple_reuse_registers_time;
	GpuTimer timer;
	
	
	//------------------------------------------------------------------------------
	//---------> Shared memory kernel
	transfer_in=0.0; transfer_out=0.0; FFT_time=0.0; FFT_external_time=0.0; FFT_multiple_time=0.0; FFT_multiple_reuse_time=0.0; cuFFT_time=0.0; FFT_multiple_reuse_registers_time=0.0;

	//---------> Memory allocation
	if (DEBUG) printf("Device memory allocation...: \t\t");
	float2 *d_output;
	float2 *d_input;
	timer.Start();
	checkCudaErrors(hipMalloc((void **) &d_input,  sizeof(float2)*input_size));
	checkCudaErrors(hipMalloc((void **) &d_output, sizeof(float2)*output_size));
	timer.Stop();
	if (DEBUG) printf("done in %g ms.\n", timer.Elapsed());

	//---------> FFT calculation
	if (DEBUG) printf("Transferring data to device...: \t");
	timer.Start();
	checkCudaErrors(hipMemcpy(d_input, h_input, input_size*sizeof(float2), hipMemcpyHostToDevice));
	timer.Stop();
	transfer_in+=timer.Elapsed();
	if (DEBUG) printf("done in %g ms.\n", timer.Elapsed());
	
	//-----> Compute FFT on the chunk
	if(CUFFT){
		//---------> FFT
		hipfftHandle plan;
		hipfftResult error;
		error = hipfftPlan1d(&plan, nSamples, HIPFFT_C2C, nSpectra);
		if (HIPFFT_SUCCESS != error){
			printf("CUFFT error: %d", error);
		}
		
		timer.Start();
		hipfftExecC2C(plan, (hipfftComplex *)d_input, (hipfftComplex *)d_output, HIPFFT_FORWARD);
		timer.Stop();
		cuFFT_time += timer.Elapsed();
		
		hipfftDestroy(plan);
	}
	
	if(MULTIPLE){
		if (DEBUG) printf("Multiple FFT...: \t\t\t");
		FFT_init();
		FFT_multiple_benchmark(d_input, d_output, nSamples, nSpectra, &FFT_multiple_time);
		if (DEBUG) printf("done in %g ms.\n", FFT_multiple_time);
	}
    
	if(EXTERNAL){
		if (DEBUG) printf("FFT...: \t\t\t\t");
		FFT_init();
		FFT_external_benchmark(d_input, d_output, nSamples, nSpectra, &FFT_external_time);
		if (DEBUG) printf("done in %g ms.\n", FFT_external_time);
	}
	
	//-----> Copy chunk of output data to host
	if (DEBUG) printf("Transferring data to host...: \t\t");
	timer.Start();
	checkCudaErrors(hipMemcpy( h_output, d_output, output_size*sizeof(float2), hipMemcpyDeviceToHost));
	timer.Stop();
	transfer_out+=timer.Elapsed();
	if (DEBUG) printf("done in %g ms.\n", timer.Elapsed());
	

	

	//---------> error check -----
	checkCudaErrors(hipGetLastError());
	
	//---------> Feeing allocated resources
	checkCudaErrors(hipFree(d_input));
	checkCudaErrors(hipFree(d_output));
	
	if (DEBUG || WRITE) printf("nSpectra:%d; nSamples:%d cuFFT:%0.3f ms; FFT:%0.3f ms; FFT external:%0.3f ms; FFT multiple:%0.3f ms;\n",nSpectra,nSamples,cuFFT_time, FFT_time, FFT_external_time, FFT_multiple_time);	
	
	if (WRITE){ 
		char str[200];
		sprintf(str,"GPU-FFT-Stockham.dat");
		if (DEBUG) printf("\n Write results into file...\t");
		save_time(str, nSpectra,nSamples, cuFFT_time, FFT_time, FFT_external_time, FFT_multiple_time, FFT_multiple_reuse_time, FFT_multiple_reuse_registers_time, transfer_in, transfer_out);
		if (DEBUG) printf("\t done.\n-------------------------------------\n");
	}
	
	return(1);
}